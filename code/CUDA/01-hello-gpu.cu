
#include <hip/hip_runtime.h>
#include <stdio.h>

void CPUFunction() {
  printf("Esta função está definida para ser executada na CPU.\n");
}

__global__ void GPUFunction() {
  printf("Esta função está definida para ser executada na GPU.\n");
}

int main() {
  CPUFunction();
  GPUFunction<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}